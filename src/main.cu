
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <thread>

template <unsigned n_ops>
__global__ void kernel(float* const ptr) {
	const unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;

	auto v = ptr[tid];

	for (unsigned i = 0; i < n_ops; i++) {
		v += 1.0f;
	}

	ptr[tid] = v;
}

void measure_launch_overhead(const unsigned grid_size) {
	constexpr unsigned block_size = 256;
	const auto array_length = block_size * grid_size;

	constexpr unsigned num_ops_A = 1u << 15;
	constexpr unsigned num_ops_B = num_ops_A + (1u << 20);

	float *ha, *da_A, *da_B;
	hipHostMalloc(&ha  , sizeof(float) * array_length, hipHostMallocDefault);
	hipMalloc    (&da_A, sizeof(float) * array_length);
	hipMalloc    (&da_B, sizeof(float) * array_length);

	// init array
	for (unsigned i = 0; i < grid_size * block_size; i++) {
		ha[i] = static_cast<float>(i);
	}
	hipMemcpy(da_A, ha, sizeof(float) * array_length, hipMemcpyDefault);
	hipMemcpy(da_B, ha, sizeof(float) * array_length, hipMemcpyDefault);

	// measure elapsed time of A ops
	const auto start_A = std::chrono::high_resolution_clock::now();
	kernel<num_ops_A><<<grid_size, block_size>>>(da_A);
	hipDeviceSynchronize();
	const auto end_A = std::chrono::high_resolution_clock::now();
	const auto time_A = std::chrono::duration_cast<std::chrono::microseconds>(end_A - start_A).count();

	// Sleep 2000ms to cool GPU
	using namespace std::chrono_literals;
	std::this_thread::sleep_for(2000ms);

	// measure elapsed time of B ops
	const auto start_B = std::chrono::high_resolution_clock::now();
	kernel<num_ops_B><<<grid_size, block_size>>>(da_B);
	hipDeviceSynchronize();
	const auto end_B = std::chrono::high_resolution_clock::now();
	const auto time_B = std::chrono::duration_cast<std::chrono::microseconds>(end_B - start_B).count();

	const auto time_diff = time_B - time_A;

	std::printf("# GridSize = %u, BlockSize = %u\n", grid_size, block_size);
	std::printf("[A] elapsed time : %lu [ns]\n", time_A);
	std::printf("[B] elapsed time : %lu [ns]\n", time_B);
	std::printf("launch overhead : %lu [ns]", time_A - time_diff * num_ops_A / (num_ops_B - num_ops_A));

	hipFree    (da_A);
	hipFree    (da_B);
	hipHostFree(ha);
}

int main() {
	for (unsigned i = 0; i < 10; i++) {
		measure_launch_overhead(1u << i);
	}
}
