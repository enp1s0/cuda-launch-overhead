
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <thread>

template <unsigned n_ops>
__global__ void kernel(float* const ptr) {
	const unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;

	auto v = ptr[tid];

	for (unsigned i = 0; i < n_ops; i++) {
		v += 1.0f;
	}

	ptr[tid] = v;
}

void measure_launch_overhead(const unsigned grid_size) {
	constexpr unsigned block_size = 1024;
	const auto array_length = block_size * grid_size;

	constexpr unsigned num_ops_A = 1u << 15;
	constexpr unsigned num_ops_B = num_ops_A + (1u << 20);

	float *ha, *da_A, *da_B;
	hipHostMalloc(&ha  , sizeof(float) * array_length, hipHostMallocDefault);
	hipMalloc    (&da_A, sizeof(float) * array_length);
	hipMalloc    (&da_B, sizeof(float) * array_length);

	// init array
	for (unsigned i = 0; i < grid_size * block_size; i++) {
		ha[i] = static_cast<float>(i);
	}
	hipMemcpy(da_A, ha, sizeof(float) * array_length, hipMemcpyDefault);
	hipMemcpy(da_B, ha, sizeof(float) * array_length, hipMemcpyDefault);

	// measure elapsed time of A ops
	const auto start_A = std::chrono::high_resolution_clock::now();
	kernel<num_ops_A><<<grid_size, block_size>>>(da_A);
	hipDeviceSynchronize();
	const auto end_A = std::chrono::high_resolution_clock::now();
	const auto time_A = std::chrono::duration_cast<std::chrono::nanoseconds>(end_A - start_A).count();

	// Sleep 2000ms to cool GPU
	using namespace std::chrono_literals;
	std::this_thread::sleep_for(2000ms);

	// measure elapsed time of B ops
	const auto start_B = std::chrono::high_resolution_clock::now();
	kernel<num_ops_B><<<grid_size, block_size>>>(da_B);
	hipDeviceSynchronize();
	const auto end_B = std::chrono::high_resolution_clock::now();
	const auto time_B = std::chrono::duration_cast<std::chrono::nanoseconds>(end_B - start_B).count();

	const auto time_diff = time_B - time_A;

	std::printf("[GridSize = %8u, BlockSize = %3u] ", grid_size, block_size);
	std::printf("elapsed time A : %15lu [ns], elapsed time B : %15lu [ns], time/op : %e [ns], launch overhead : %7lu [ns]\n",
			time_A,
			time_B,
			static_cast<double>(time_diff) / (num_ops_B - num_ops_A),
			time_A - time_diff * num_ops_A / (num_ops_B - num_ops_A));

	hipFree    (da_A);
	hipFree    (da_B);
	hipHostFree(ha);
}

int main() {
	for (unsigned i = 0; i < 20; i++) {
		measure_launch_overhead(1u << i);
	}
}
